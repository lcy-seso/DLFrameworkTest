#include "hip/hip_runtime.h"

#include <iomanip>
#include <iostream>

#include "cutlass/aligned_buffer.h"
#include "cutlass/core_io.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/matrix_shape.h"
#include "cutlass/numeric_types.h"
#include "cutlass/transform/pitch_linear_thread_map.h"
#include "cutlass/transform/threadblock/predicated_tile_iterator.h"
#include "cutlass/transform/threadblock/regular_tile_iterator_tensor_op.h"
#include "cutlass/util/debug.h"
#include "cutlass/util/device_dump.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

template <typename Element, typename GmemIterator, typename SmemIterator,
          const int m, const int n>
__global__ void test_kernel_dump(typename GmemIterator::Params params,
                                 Element* data) {
  extern __shared__ Element shared_storage[];

  // Construct the global iterator and load the data to the fragments.
  int tb_thread_id = threadIdx.y * blockDim.x + threadIdx.x;

  GmemIterator gmem_iterator(params, data, {m, n}, tb_thread_id);

  typename GmemIterator::Fragment frag;
  frag.clear();
  gmem_iterator.load(frag);

  SmemIterator smem_iterator(
      typename SmemIterator::TensorRef(
          {shared_storage, SmemIterator::Layout::packed({m, n})}),
      tb_thread_id);

  smem_iterator.store(frag);

  // Call dump_shmem() with different parameters.
  if (threadIdx.x == 0 && blockIdx.x == 0) printf("\nDump all the elements:\n");
  cutlass::debug::dump_shmem(shared_storage, m * n);
}

int main() {
  const int m = 8;
  const int n = 64;

  using Element = cutlass::half_t;
  using Layout = cutlass::layout::RowMajor;
  // using Layout = cutlass::layout::ColumnMajor;

  cutlass::HostTensor<Element, Layout> matrix({m /*ld*/, n /*strided*/});
  cutlass::reference::host::BlockFillSequential(matrix.host_data(),
                                                matrix.capacity());
  // Dump the matrix.
  // std::cout << "Matrix:\n" << matrix.host_view() << "\n";

  // Copy the matrix to the device.
  matrix.sync_device();

  dim3 grid(1, 1);
  dim3 block(32, 1, 1);

  int smem_size = int(sizeof(Element) * m * n);

  // Define a global iterator, a shared iterator and their thread map.
  using ThreadMap = cutlass::transform::PitchLinearWarpRakedThreadMap<
      cutlass::layout::PitchLinearShape<n, m>, 32 /*threads*/,
      cutlass::layout::PitchLinearShape<8, 4> /*warp arrangement*/,
      8 /*ElementPerAccess*/>;
  using GmemIterator = cutlass::transform::threadblock::PredicatedTileIterator<
      cutlass::MatrixShape<m, n>, Element, Layout, 1 /*AdvanceRank*/,
      ThreadMap>;

  typename GmemIterator::Params params(Layout::packed({m, n}));
  const int crosswise = 64;
  using SLayout = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<Element>::value, crosswise>;
  // using SLayout = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
  //     cutlass::sizeof_bits<Element>::value, crosswise>;
  using SmemIterator = cutlass::transform::threadblock::RegularTileIterator<
      cutlass::MatrixShape<m, n>, Element, SLayout, 1, ThreadMap>;

  test_kernel_dump<Element /*element type*/, GmemIterator /*source iterator*/,
                   SmemIterator /*target iterator*/, m, n>
      <<<grid, block, smem_size, 0>>>(params, matrix.device_ref().data());

  hipError_t result = hipDeviceSynchronize();

  if (result != hipSuccess) {
    std::cout << "Failed" << std::endl;
  }

  return (result == hipSuccess ? 0 : -1);

  return 0;
}
