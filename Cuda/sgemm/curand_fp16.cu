#include "hip/hip_runtime.h"

#include <stdexcept>

#include "curand_fp16.h"

namespace {
constexpr unsigned block_size = 256;
constexpr unsigned store_block_batch_size = 1;
constexpr unsigned num_sm_scale = 1;
using block_t = ulong1;

template <class T>
__global__ void status_init_kernel(T* const status_ptr,
                                   const std::uint64_t seed) {
  const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  hiprand_init(seed, tid, 0, status_ptr + tid);
}

template <class T>
struct size_of {
  int value = 0;
};
template <>
struct size_of<ushort1> {
  static const int value = 2;
};
template <>
struct size_of<uint1> {
  static const int value = 4;
};
template <>
struct size_of<ulong1> {
  static const int value = 8;
};
template <>
struct size_of<half> {
  static const int value = 2;
};
template <>
struct size_of<half2> {
  static const int value = 4;
};
// template <> struct size_of<ulong2 > {static const int value = 16;};

template <class RNG_T, class BLOCK_T, uint32_t pm = 0>
__global__ void generate_kernel(half* const array_ptr, RNG_T* const status_ptr,
                                const std::size_t size) {
  constexpr auto batch_size =
      size_of<BLOCK_T>::value / size_of<half>::value * store_block_batch_size;
  const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  auto curand_gen = *(status_ptr + tid);

  const auto batch_loop_size = size - (size % batch_size);
  for (unsigned i = tid * batch_size; i < batch_loop_size;
       i += batch_size * gridDim.x * blockDim.x) {
    // block gen
    union {
      half h1[size_of<BLOCK_T>::value / size_of<half>::value];
      half2 h2[size_of<BLOCK_T>::value / size_of<half2>::value];
      BLOCK_T store_block;
      unsigned u[size_of<BLOCK_T>::value / size_of<uint1>::value];
      short s[size_of<BLOCK_T>::value / size_of<ushort1>::value];
    } batch_block[store_block_batch_size];

#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
#pragma unroll
      for (unsigned j = 0; j < size_of<BLOCK_T>::value / size_of<half2>::value;
           j++) {
        batch_block[sb].u[j] = hiprand(&curand_gen);
        if constexpr (pm == 0) {
          batch_block[sb].u[j] &= 0x7fff7fffu;
        }
      }
    }
#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
#pragma unroll
      for (unsigned j = 0; j < size_of<BLOCK_T>::value / size_of<half>::value;
           j++) {
        batch_block[sb].h1[j] = __short2half_rn(batch_block[sb].s[j]);
      }
    }
#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
#pragma unroll
      for (unsigned j = 0; j < size_of<BLOCK_T>::value / size_of<half2>::value;
           j++) {
        batch_block[sb].h2[j] =
            __hmul2(batch_block[sb].h2[j], __float2half2_rn(1.f / 0x7fff));
      }
    }
#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
      *(reinterpret_cast<BLOCK_T*>(array_ptr + i) + sb) =
          batch_block[sb].store_block;
    }
  }
  if (tid == 0) {
    const auto res = size - batch_loop_size;
    if (res != 0) {
      for (unsigned j = 0; j < res; j++) {
        const auto v =
            static_cast<short>(hiprand(&curand_gen) & (pm ? 0xffff : 0x7fff));
        array_ptr[batch_loop_size + j] =
            __hmul(__short2half_rn(v), __float2half_rn(1.f / 0x7fff));
      }
    }
  }
  *(status_ptr + tid) = curand_gen;
}

template <class RNG_T, class BLOCK_T>
__global__ void generate_normal_kernel(half* const array_ptr,
                                       RNG_T* const status_ptr,
                                       const std::size_t size, const float mean,
                                       const float sigma) {
  constexpr auto batch_size =
      size_of<BLOCK_T>::value / size_of<half>::value * store_block_batch_size;
  const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  auto curand_gen = *(status_ptr + tid);

  const auto batch_loop_size = size - (size % batch_size);
  for (unsigned i = tid * batch_size; i < batch_loop_size;
       i += batch_size * gridDim.x * blockDim.x) {
    // block gen
    union {
      half h1[size_of<BLOCK_T>::value / size_of<half>::value];
      half2 h2[size_of<BLOCK_T>::value / size_of<half2>::value];
      BLOCK_T store_block;
      unsigned u[size_of<BLOCK_T>::value / size_of<uint1>::value];
      short s[size_of<BLOCK_T>::value / size_of<ushort1>::value];
      ushort2 s2[size_of<BLOCK_T>::value / size_of<uint1>::value];
    } batch_block[store_block_batch_size];

#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
#pragma unroll
      for (unsigned j = 0; j < size_of<BLOCK_T>::value / size_of<half2>::value;
           j++) {
        batch_block[sb].u[j] = hiprand(&curand_gen);
        batch_block[sb].u[j] |= 0x00010001u;
      }
    }
#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
#pragma unroll
      for (unsigned j = 0; j < size_of<BLOCK_T>::value / size_of<uint1>::value;
           j++) {
        const auto s2 = batch_block[sb].s2[j];
        const auto f2_x = static_cast<float>(s2.x) / 0x10000;
        const auto mag = sigma * sqrtf(-2.0f * logf(f2_x));
        constexpr float two_pi = M_PI * 2;
        const auto f2_y = static_cast<float>(s2.y) / 0x10000;
        float si, co;
        sincosf(two_pi * f2_y, &si, &co);
        const auto h2 = __float22half2_rn(make_float2(si, co));
        batch_block[sb].h2[j] =
            __hfma2(__float2half2_rn(mag), h2, __float2half2_rn(mean));
      }
    }
#pragma unroll
    for (unsigned sb = 0; sb < store_block_batch_size; sb++) {
      *(reinterpret_cast<BLOCK_T*>(array_ptr + i) + sb) =
          batch_block[sb].store_block;
    }
  }
  if (tid == 0) {
    const auto res = size - batch_loop_size;
    if (res != 0) {
      for (unsigned j = 0; j < res; j++) {
        const auto v = (hiprand_normal(&curand_gen) + mean) * sigma;
        array_ptr[batch_loop_size + j] = __float2half(v);
      }
    }
  }
  *(status_ptr + tid) = curand_gen;
}
}  // namespace

void curand_fp16::create(generator_t& gen, const hiprandRngType_t rng_type) {
  // set cuda stream
  gen.hip_stream = 0;
  // get num sm
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  gen.num_sm = prop.multiProcessorCount;

  // calculate grid_size
  gen.num_threads = gen.num_sm * num_sm_scale * block_size;

  // set algo
  gen.rng_type = rng_type;

  // set generator
  unsigned state_struct_size = 0;
  switch (rng_type) {
#define CASE_RNG_TYPE(rng)                                        \
  case rng:                                                       \
    state_struct_size =                                           \
        sizeof(typename curand_fp16::curand_status_t<rng>::type); \
    break
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A);
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW);
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    default:
      throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
  }
  const auto stat =
      hipMalloc(&gen.status_ptr, state_struct_size * gen.num_threads);
  if (stat != hipSuccess) {
    throw std::runtime_error(
        "[curand_fp16 error] : " + std::string(hipGetErrorString(stat)) +
        " @" + __func__);
  }
}

void curand_fp16::set_seed(generator_t& gen, const std::uint64_t seed) {
  switch (gen.rng_type) {
#define CASE_RNG_TYPE(rng)                                                  \
  case rng:                                                                 \
    status_init_kernel<typename curand_fp16::curand_status_t<rng>::type>    \
        <<<gen.num_threads / block_size, block_size, 0, gen.hip_stream>>>( \
            reinterpret_cast<                                               \
                typename curand_fp16::curand_status_t<rng>::type*>(         \
                gen.status_ptr),                                            \
            seed);                                                          \
    break
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A);
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW);
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    default:
      throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
  }
}

void curand_fp16::uniform(generator_t& gen, half* const ptr,
                          const std::size_t size, const bool pm) {
  const auto batch_size = size_of<block_t>::value / size_of<half>::value;
  const auto grid_size = std::min<unsigned>(
      std::min<unsigned>(gen.num_threads / block_size,
                         (size + block_size - 1) / block_size),
      ((size + batch_size - 1) / batch_size + block_size - 1) / block_size);
  if (pm == 0) {
    switch (gen.rng_type) {
#define CASE_RNG_TYPE(rng)                                                     \
  case rng:                                                                    \
    generate_kernel<typename curand_fp16::curand_status_t<rng>::type, block_t, \
                    0><<<grid_size, block_size, 0, gen.hip_stream>>>(         \
        ptr,                                                                   \
        reinterpret_cast<typename curand_fp16::curand_status_t<rng>::type*>(   \
            gen.status_ptr),                                                   \
        size);                                                                 \
    break
      CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A);
      CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW);
      CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
      default:
        throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
    }
  } else {
    switch (gen.rng_type) {
#define CASE_RNG_TYPE(rng)                                                     \
  case rng:                                                                    \
    generate_kernel<typename curand_fp16::curand_status_t<rng>::type, block_t, \
                    1><<<grid_size, block_size, 0, gen.hip_stream>>>(         \
        ptr,                                                                   \
        reinterpret_cast<typename curand_fp16::curand_status_t<rng>::type*>(   \
            gen.status_ptr),                                                   \
        size);                                                                 \
    break
      CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A);
      CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW);
      CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
      default:
        throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
    }
  }
}

void curand_fp16::normal(generator_t& gen, half* const ptr,
                         const std::size_t size, const float mean,
                         const float var) {
  const auto batch_size = size_of<block_t>::value / size_of<half>::value;
  const auto grid_size = std::min<unsigned>(
      std::min<unsigned>(gen.num_threads / block_size,
                         (size + block_size - 1) / block_size),
      ((size + batch_size - 1) / batch_size + block_size - 1) / block_size);
  switch (gen.rng_type) {
#define CASE_RNG_TYPE(rng)                                                   \
  case rng:                                                                  \
    generate_normal_kernel<                                                  \
        typename curand_fp16::curand_status_t<rng>::type,                    \
        block_t><<<grid_size, block_size, 0, gen.hip_stream>>>(             \
        ptr,                                                                 \
        reinterpret_cast<typename curand_fp16::curand_status_t<rng>::type*>( \
            gen.status_ptr),                                                 \
        size, mean, std::sqrt(var));                                         \
    break
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A);
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW);
    CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
    default:
      throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
  }
}

void curand_fp16::set_cuda_stream(curand_fp16::generator_t& generator,
                                  hipStream_t const hip_stream) {
  generator.hip_stream = hip_stream;
}

void curand_fp16::destroy(generator_t& gen) { hipFree(gen.status_ptr); }
