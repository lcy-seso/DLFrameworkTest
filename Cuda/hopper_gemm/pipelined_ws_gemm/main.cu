#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include "gemm.cuh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <iomanip>
#include <iostream>
#include <stdexcept>

template <typename DType, const int kM_, const int kN_, const int kK_,
          const int kTM_, const int kTN_, const int kTK_,
          const int kNumStages_ = 1, const int kNumTmaMulticast_ = 1>
struct GemmTraits {
  static constexpr int kM = kM_;
  static constexpr int kN = kN_;
  static constexpr int kK = kK_;

  static constexpr int kTM = kTM_;
  static constexpr int kTN = kTN_;
  static constexpr int kTK = kTK_;

  static constexpr int kNumTmaMulticast = kNumTmaMulticast_;

  static constexpr int kNumStages = kNumStages_;

  // the size of shared memory for each operand and result
  static constexpr int kSizeA = kTM * kTK * sizeof(DType);
  static constexpr int kSizeB = kTN * kTK * sizeof(DType);
  static constexpr int kSizeC = kTM * kTN * sizeof(DType);

  static_assert(kSizeC % 1024 == 0,
                "Shared memory of output tensor must be aligned to 1024 bytes");
  static_assert(kSizeA % 1024 == 0,
                "Shared memory of operand A must be aligned to 1024 bytes");
  static_assert(kSizeB % 1024 == 0,
                "Shared memory of operand B must be aligned to 1024 bytes");

  static constexpr int kSharedDataSize =
      (kSizeA + kSizeB) * kNumStages + kSizeC;
  static constexpr int kSharedMemSize =  // data + barriers
      kSharedDataSize + kNumStages * sizeof(uint64_t) * 2;

  static constexpr int kExpectedTmaBytes = kSizeA + kSizeB;

  static constexpr uint32_t kKShapeAllStages = kNumStages * kTK;

  static_assert(kK % kKShapeAllStages == 0,
                "kK must be divisible by kKShapeAllStages");

  static constexpr uint32_t kKNumIterations = CeilDiv<kK, kKShapeAllStages>;

  static constexpr uint32_t kNumWarpGroup = 2;
  static constexpr uint32_t kThreads = 128 * kNumWarpGroup;
  // thread 0 ~ kMathThreads - 1: consumer
  // thread kMathThreads ~ kThreads - 1: producer
  static constexpr uint32_t kMathThreads = 128;

  // register reconfigurations
  static constexpr uint32_t kNumTMARegisters = 40;
  static constexpr uint32_t kNumMathRegisters = 232;

  // tile scheduler
  using Scheduler_ = Scheduler<kM, kN, kTM, kTN, kNumTmaMulticast>;
};

int main() {
  //// kernel parameters
  using DType = __hip_bfloat16;
  // using DType = __hip_fp8_e4m3_fnuz;

  static constexpr uint64_t kM = 64;
  static constexpr uint64_t kN = 64;
  static constexpr uint64_t kK = 128;

  static constexpr uint64_t kTM = 64;
  static constexpr uint64_t kTN = 64;
  static constexpr uint64_t kTK = 64;

  static constexpr int kNumStages = 1;
  using Traits = GemmTraits<DType, kM, kN, kK, kTM, kTN, kTK, kNumStages>;

  /// create data
  thrust::host_vector<DType> h_a(kM * kK);
  thrust::host_vector<DType> h_b(kK * kN);
  thrust::host_vector<DType> h_c(kM * kN);

  for (int i = 0; i < h_a.size(); ++i) {
    h_a[i] = static_cast<DType>(i % 256);
    // h_a[i] = static_cast<DType>(rand_float());
  }
  thrust::device_vector<DType> d_a = h_a;

  // print_values<DType, kM, kK>(h_a.data(), 0, kM * kK);

  for (int i = 0; i < h_b.size(); ++i) {
    h_b[i] = static_cast<DType>(i % 256);
    // h_b[i] = static_cast<DType>(rand_float());
  }
  thrust::device_vector<DType> d_b = h_b;

  thrust::fill(h_c.begin(), h_c.end(), static_cast<DType>(0));
  thrust::device_vector<DType> d_c = h_c;
  CHECK_CUDA(hipDeviceSynchronize());

  //// create TMA descriptors
  // operand A is laid out in row-major order
  TMADescriptor<DType> tma_desc_a;
  uint64_t global_dim_a[2] = {kK, kM};
  uint32_t shared_dim_a[2] = {kTK, kTM};
  tma_desc_a.create_tma_2d_desc(
      thrust::raw_pointer_cast(d_a.data()),  // Global address
      global_dim_a,                          // Global dimensions
      shared_dim_a,  // Shared memory dimensions (box dimensions)
      kK             // Global stride in bytes
  );

  // operand B is laid out in column-major order
  TMADescriptor<DType> tma_desc_b;
  uint64_t global_dim_b[2] = {kK, kN};
  uint32_t shared_dim_b[2] = {kTK, kTN};
  tma_desc_b.create_tma_2d_desc(
      thrust::raw_pointer_cast(d_b.data()),  // Global address
      global_dim_b,                          // Global dimensions
      shared_dim_b,  // Shared memory dimensions (box dimensions)
      kK             // Global stride in bytes
  );

  // operand C is laid out in row-major order
  TMADescriptor<DType> tma_desc_c;
  uint64_t global_dim_c[2] = {kN, kM};
  uint32_t shared_dim_c[2] = {kTN, kTM};
  tma_desc_c.create_tma_2d_desc(
      thrust::raw_pointer_cast(d_c.data()),  // Global address
      global_dim_c,                          // Global dimensions
      shared_dim_c,  // Shared memory dimensions (box dimensions)
      kN             // Global stride in bytes
  );

  //// launch kernel
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  uint32_t num_sms = deviceProp.multiProcessorCount;

  dim3 blocks(num_sms, 1);
  dim3 threads(Traits::kThreads, 1, 1);

  std::cout << "num_sms: " << num_sms << std::endl;
  std::cout << "threads: " << threads.x << std::endl;
  std::cout << "shared memory size: " << Traits::kSharedMemSize << std::endl;
  std::cout << "shared memory per block: " << deviceProp.sharedMemPerBlock
            << std::endl
            << std::endl;

  auto kernel = &hopper_gemm<DType, Traits>;

  if (Traits::kSharedMemSize > GetMaxSharedMemoryPerBlock()) {
    CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel),
                                    hipFuncAttributeMaxDynamicSharedMemorySize,
                                    Traits::kSharedMemSize));
  }

  kernel<<<blocks, threads, Traits::kSharedMemSize>>>(
      tma_desc_a.get_tma_desc(), tma_desc_b.get_tma_desc(),
      tma_desc_c.get_tma_desc());

  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
  return 0;
}
