#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include "gemm.cuh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <iomanip>
#include <iostream>
#include <stdexcept>

template <typename DType, const int kM_, const int kN_, const int kK_,
          const int kTM_, const int kTN_, const int kTK_,
          const int kNumStages_ = 1, const int kNumTmaMulticast_ = 1>
struct GemmTraits {
  static constexpr int kM = kM_;
  static constexpr int kN = kN_;
  static constexpr int kK = kK_;

  static constexpr int kTM = kTM_;
  static constexpr int kTN = kTN_;
  static constexpr int kTK = kTK_;

  static constexpr int kNumTmaMulticast = kNumTmaMulticast_;

  static constexpr int kNumStages = kNumStages_;

  static constexpr int kShapeA = kTM * kTK;
  static constexpr int kShapeB = kTK * kTN;
  static constexpr int kShapeC = kTM * kTN;

  // the size of shared memory for each operand and result
  static constexpr int kSizeA = kShapeA * sizeof(DType);
  static constexpr int kSizeB = kShapeB * sizeof(DType);
  static constexpr int kSizeC = kShapeC * sizeof(DType);

  static_assert(kSizeC % 1024 == 0,
                "Shared memory of output tensor must be aligned to 1024 bytes");
  static_assert(kSizeA % 1024 == 0,
                "Shared memory of operand A must be aligned to 1024 bytes");
  static_assert(kSizeB % 1024 == 0,
                "Shared memory of operand B must be aligned to 1024 bytes");

  static constexpr int kSharedDataSize =
      (kSizeA + kSizeB) * kNumStages + kSizeC;
  static constexpr int kSharedMemSize =  // data + barriers
      kSharedDataSize + kNumStages * sizeof(uint64_t) * 2;

  static constexpr int kExpectedTmaBytes = kSizeA + kSizeB;

  static constexpr uint32_t kKShapeAllStages = kNumStages * kTK;

  static_assert(kK % kKShapeAllStages == 0,
                "kK must be divisible by kKShapeAllStages");

  static constexpr uint32_t kKNumIterations = CEIL_DIV(kK, kKShapeAllStages);

  static constexpr uint32_t kNumWarpGroup = 2;
  static constexpr uint32_t kThreads = 128 * kNumWarpGroup;
  // thread 0 ~ kMathThreads - 1: consumer
  // thread kMathThreads ~ kThreads - 1: producer
  static constexpr uint32_t kMathThreads = 128;

  // register reconfigurations
  static constexpr uint32_t kNumTMARegisters = 40;
  static constexpr uint32_t kNumMathRegisters = 232;

  // tile scheduler
  using Scheduler_ = Scheduler<kM, kN, kTM, kTN, kNumTmaMulticast>;
};

int main() {
  //// kernel parameters
  using DType = __hip_bfloat16;
  // using DType = __hip_fp8_e4m3_fnuz;

  static constexpr uint64_t kM = 640;
  static constexpr uint64_t kN = 4096;
  static constexpr uint64_t kK = 1280;

  static constexpr uint64_t kTM = 64;
  static constexpr uint64_t kTN = 64;
  static constexpr uint64_t kTK = 64;

  static constexpr int kNumStages = 4;
  using Traits = GemmTraits<DType, kM, kN, kK, kTM, kTN, kTK, kNumStages>;

  /// create data
  thrust::host_vector<DType> h_a(kM * kK);
  thrust::host_vector<DType> h_b(kK * kN);
  thrust::host_vector<DType> h_c(kM * kN);

  for (int i = 0; i < h_a.size(); ++i) {
    h_a[i] = static_cast<DType>(i % 256);
    // h_a[i] = static_cast<DType>(rand_float());
  }
  thrust::device_vector<DType> d_a = h_a;

  for (int i = 0; i < h_b.size(); ++i) {
    // Initialize matrix B in column-major order
    // For column-major: element (i,j) is at index i + j * kK
    int row = i % kK;
    int col = i / kK;
    h_b[i] = static_cast<DType>((row + col * kK) % 256);
  }
  thrust::device_vector<DType> d_b = h_b;

  thrust::fill(h_c.begin(), h_c.end(), static_cast<DType>(0));
  thrust::device_vector<DType> d_c = h_c;
  CudaCheck(hipDeviceSynchronize());

  //// create TMA descriptors
  // operand A is laid out in row-major order
  TMADescriptor<DType> tma_desc_a;
  uint64_t global_dim_a[2] = {kK, kM};
  uint32_t shared_dim_a[2] = {kTK, kTM};
  tma_desc_a.create_tma_2d_desc(
      thrust::raw_pointer_cast(d_a.data()),  // Global address
      global_dim_a,                          // Global dimensions
      shared_dim_a,  // Shared memory dimensions (box dimensions)
      kK             // Global stride in bytes
  );

  // operand B is laid out in column-major order
  TMADescriptor<DType> tma_desc_b;
  uint64_t global_dim_b[2] = {kK, kN};
  uint32_t shared_dim_b[2] = {kTK, kTN};
  tma_desc_b.create_tma_2d_desc(
      thrust::raw_pointer_cast(d_b.data()),  // Global address
      global_dim_b,                          // Global dimensions
      shared_dim_b,  // Shared memory dimensions (box dimensions)
      kK             // Global stride in bytes (distance between columns)
  );

  // operand C is laid out in row-major order
  TMADescriptor<DType> tma_desc_c;
  uint64_t global_dim_c[2] = {kN, kM};
  uint32_t shared_dim_c[2] = {kTN, kTM};
  tma_desc_c.create_tma_2d_desc(
      thrust::raw_pointer_cast(d_c.data()),  // Global address
      global_dim_c,                          // Global dimensions
      shared_dim_c,  // Shared memory dimensions (box dimensions)
      kN             // Global stride in bytes
  );

  //// launch kernel
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  uint32_t num_sms = deviceProp.multiProcessorCount;

  dim3 blocks(num_sms, 1);
  dim3 threads(Traits::kThreads, 1, 1);

  std::cout << "num_sms: " << num_sms << std::endl;
  std::cout << "threads: " << threads.x << std::endl;
  std::cout << "shared memory size: " << Traits::kSharedMemSize << std::endl;
  std::cout << "shared memory per block: " << deviceProp.sharedMemPerBlock
            << std::endl
            << std::endl;

  auto kernel = &hopper_gemm<DType, Traits>;

  if (Traits::kSharedMemSize > GetMaxSharedMemoryPerBlock()) {
    CudaCheck(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel),
                                   hipFuncAttributeMaxDynamicSharedMemorySize,
                                   Traits::kSharedMemSize));
  }

  kernel<<<blocks, threads, Traits::kSharedMemSize>>>(
      tma_desc_a.get_tma_desc(), tma_desc_b.get_tma_desc(),
      tma_desc_c.get_tma_desc());

  CudaCheck(hipGetLastError());
  CudaCheck(hipDeviceSynchronize());
  return 0;
}
