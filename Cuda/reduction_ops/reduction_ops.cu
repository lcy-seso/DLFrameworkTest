#include "reduction_kernel.h"

int main(int argc, char* argv[]) {
  std::vector<int> kTensorShape = {13, 33, 11};
  std::vector<int> axes = {1};
  int out_rank = 0;

  int product = std::accumulate(kTensorShape.begin(), kTensorShape.end(), 1,
      std::multiplies<int>());
  printf("reduce %d numbers.\n", product);

  const int kMaxThreads = 512;
  const int kMaxBlocks = 64;

  srand(0);
  float *h_a, *h_b;

  hipHostMalloc((void**)&h_a, sizeof(float) * product);
  // TODO(ying) The current output size is the same as the input which is not
  // correct.
  hipHostMalloc((void**)&h_b, sizeof(float) * product);

  // random initialization of matrix A.
  for (size_t i = 0; i < product; ++i) h_a[i] = static_cast<float>(i + 1);

  // initialize memory that stores computation result to all zeros;
  memset(h_b, 0., sizeof(float) * product);

  // events to count the execution time.
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate memory space on the device.
  float *d_a, *d_b;
  hipMalloc((void**)&d_a, sizeof(float) * product);
  hipMalloc((void**)&d_b, sizeof(float) * product);

  // copy matrix A from host to device memory
  CHECK(hipMemcpy(d_a, h_a, sizeof(float) * product, hipMemcpyHostToDevice));

  // start to count execution time. use the default stream.
  hipEventRecord(start);

  // lanuch kernel.
  int in_dim0 = kTensorShape[0];
  int in_dim1 = kTensorShape.size() > 1 ? kTensorShape[1] : 1;
  int in_dim2 = kTensorShape.size() > 2 ? kTensorShape[2] : 1;
  // ReduceImpl<float, Sum<float>>(d_a, d_b, axes, kTensorShape.size(), in_dim0,
  //                               in_dim1, in_dim2, out_rank, Sum<float>(),
  //                               kMaxThreads, kMaxBlocks);
  ReduceImpl<float, Max<float>>(d_a, d_b, axes, kTensorShape.size(), in_dim0,
      in_dim1, in_dim2, out_rank, Max<float>(),
      kMaxThreads, kMaxBlocks);

  hipEventRecord(stop);
  CHECK(hipEventSynchronize(stop));
  CHECK(hipMemcpy(h_b, d_b, sizeof(float) * product, hipMemcpyDeviceToHost));

  float kernel_elapsed_time;
  hipEventElapsedTime(&kernel_elapsed_time, start, stop);
  printf("kernel execution time elapse : %f\n", kernel_elapsed_time);

  printf("reduced sum = %.2f\n", h_b[0]);

  hipFree(d_a);
  hipFree(d_b);
  hipHostFree(h_a);
  hipHostFree(h_b);

  return 0;
}
