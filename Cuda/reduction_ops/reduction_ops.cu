#include "reduction_kernel.h"
#include "reduction_ops.h"

int main(int argc, char* argv[]) {
  std::vector<size_t> kTensorShape = {23, 13, 11};
  size_t product = std::accumulate(kTensorShape.begin(), kTensorShape.end(), 1,
                                   std::multiplies<size_t>());
  printf("reduce %d numbers.\n", static_cast<int>(product));

  const int kMaxThreads = 512;
  const int kMaxBlocks = 64;

  srand(0);

  float *h_a, *h_b;

  hipHostMalloc((void**)&h_a, sizeof(float) * product);
  hipHostMalloc((void**)&h_b, sizeof(float) * product);

  // random initialization of matrix A.
  for (size_t i = 0; i < product; ++i) h_a[i] = static_cast<float>(i + 1);

  // initialize memory that stores computation result to all zeros;
  memset(h_b, 0., sizeof(float) * product);

  // events to count the execution time.
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate memory space on the device.
  float *d_a, *d_b;
  hipMalloc((void**)&d_a, sizeof(float) * product);
  hipMalloc((void**)&d_b, sizeof(float) * product);

  // copy matrix A from host to device memory
  CHECK(hipMemcpy(d_a, h_a, sizeof(float) * product, hipMemcpyHostToDevice));

  // start to count execution time. use the default stream.
  hipEventRecord(start);

  // lanuch kernel.
  int numBlocks = 0;
  int numThreads = 0;
  getNumBlocksAndThreads(product, kMaxBlocks, kMaxThreads, numBlocks,
                         numThreads);
  printf("numThreads = %d, numBlocks = %d\n", numThreads, numBlocks);

  reduceToScalar(numThreads, numBlocks, product, d_a, d_b);

  hipEventRecord(stop);
  CHECK(hipEventSynchronize(stop));
  CHECK(hipMemcpy(h_b, d_b, sizeof(float) * product, hipMemcpyDeviceToHost));

  float kernel_elapsed_time;
  hipEventElapsedTime(&kernel_elapsed_time, start, stop);
  printf("kernel execution time elapse : %f\n", kernel_elapsed_time);

  printf("reduced sum = %.2f\n", h_b[0]);

  hipFree(d_a);
  hipFree(d_b);
  hipHostFree(h_a);
  hipHostFree(h_b);

  return 0;
}
