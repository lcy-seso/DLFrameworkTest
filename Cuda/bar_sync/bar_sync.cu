#include "hip/hip_runtime.h"
#include "barrier.cuh"
#include "copy.cuh"
#include "cuda_utils.cuh"
#include "sync.cuh"
#include "utils.cuh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cstdint>
#include <iostream>

using namespace copy;
using namespace barrier;
namespace {

__forceinline__ __device__ uint32_t smem_ptr_to_uint(void const* const ptr) {
  return static_cast<uint32_t>(__cvta_generic_to_shared(ptr));
}

__forceinline__ __device__ void fence_proxy_async() {
  asm volatile("fence.proxy.async.shared::cta;" : :);
}

__forceinline__ __device__ void mbarrier_cp_async_arrive(
    uint64_t& smem_barrier) {
  uint32_t smem_int_ptr = smem_ptr_to_uint(&smem_barrier);
  asm volatile("cp.async.mbarrier.arrive.shared.b64 [%0];"
               :
               : "r"(smem_int_ptr));
}

__device__ __forceinline__ void init_barrier(uint64_t* barrier,
                                             int arrive_count) {
  uint32_t barrier_ptr =
      static_cast<uint32_t>(__cvta_generic_to_shared(barrier));
  asm volatile(
      "{\n\t"
      "mbarrier.init.shared::cta.b64 [%1], %0; \n"
      "}"
      :
      : "r"(arrive_count), "r"(barrier_ptr));
}

__device__ __forceinline__ void arrive(uint64_t const* barrier) {
  uint32_t barrier_ptr =
      static_cast<uint32_t>(__cvta_generic_to_shared(barrier));
  asm volatile(
      "{\n\t"
      "mbarrier.arrive.shared::cta.b64 _, [%0];\n\t"  // "_" sink symbol is used
      "}"
      :
      : "r"(barrier_ptr));
}

__device__ __forceinline__ void wait(uint64_t* barrier, int phase) {
  uint32_t barrier_ptr =
      static_cast<uint32_t>(__cvta_generic_to_shared(barrier));
  constexpr uint32_t ticks = 0x989680;  // timeout
  asm volatile(
      "{\n\t"
      ".reg .pred       P1; \n\t"  // predicate register
      "LAB_WAIT: \n\t"             // spin-wait loop
      "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1, %2; \n\t"
      "@P1 bra DONE; \n\t"
      "bra     LAB_WAIT; \n\t"
      "DONE: \n\t"
      "}"
      :
      : "r"(barrier_ptr), "r"(phase), "r"(ticks));
}

template <const int kNumel>
__global__ void test_bar_sync(const bfloat16* scores, const bfloat16* bias,
                              bfloat16* output) {
  if (blockIdx.x) return;

  using DType = bfloat16;
  extern __shared__ __align__(1024) uint8_t buf_[];
  auto sptr_uint = [](auto* ptr) { return __cvta_generic_to_shared(ptr); };
  int tid = threadIdx.x;

  DType* scores_s = reinterpret_cast<DType*>(buf_);
  DType* bias_s = scores_s + kNumel;
  DType* biased_scores_s = bias_s + kNumel;

  // for barrier
  auto barriers = reinterpret_cast<uint64_t*>(biased_scores_s + kNumel);
  if (threadIdx.x == 0) {
    init_barrier(&barriers[0], 1);
    init_barrier(&barriers[1], 4);
  }
  __syncthreads();

  static constexpr int kBytesPerAccess = 128 / 8;
  static constexpr int kNumPerAccess = kBytesPerAccess / sizeof(DType);

  // step1: load scores and bias to shared memory
  int num_load_threads = kNumel / kNumPerAccess;
  int offset = tid * kNumPerAccess;

  if (tid < num_load_threads) {
    ld_global_st_shared<kBytesPerAccess>(sptr_uint(scores_s + offset),
                                         scores + offset);
    ld_global_st_shared<kBytesPerAccess>(sptr_uint(bias_s + offset),
                                         bias + offset);
  }

  // block::copy_async();
  wait(&barriers[0], 0);

  // step2 : add bias to scores using vectorized addition(128 threads)
  int compute_threads = kNumel / 2;
  if (tid < compute_threads) {
    biased_scores_s[tid] = scores_s[tid] + bias_s[tid];
    bfloat162 v_scores = *reinterpret_cast<bfloat162*>(scores_s + tid * 2);
    bfloat162 v_bias = *reinterpret_cast<bfloat162*>(bias_s + tid * 2);
    bfloat162 biased_score = __hadd2(v_scores, v_bias);

    *reinterpret_cast<bfloat162*>(biased_scores_s + tid * 2) = biased_score;

    if (tid % 32 == 0) {
      arrive(&barriers[1]);
    }
  }
  wait(&barriers[1], 0);

  // step3: store biased scores to global memory
  if (tid < num_load_threads) {
    ld_shared_st_global<kBytesPerAccess>(output + offset,
                                         sptr_uint(biased_scores_s + offset));
  }
}
}  // namespace

int main(int argc, char** argv) {
  printf("Starting program...\n");
  using DType = bfloat16;
  static constexpr int kN = 256;

  thrust::host_vector<DType> h_a(kN);
  thrust::host_vector<DType> h_b(kN);
  thrust::host_vector<DType> h_c_ref(kN);

  for (int i = 0; i < h_a.size(); ++i) {
    h_a[i] = rand_bfloat16();
    h_b[i] = rand_bfloat16();
    h_c_ref[i] = h_a[i] + h_b[i];
  }
  thrust::device_vector<DType> d_a = h_a;
  thrust::device_vector<DType> d_b = h_b;

  thrust::host_vector<DType> h_c(kN);
  thrust::fill(h_c.begin(), h_c.end(), static_cast<DType>(0));
  thrust::device_vector<DType> d_c = h_c;
  CudaCheck(hipDeviceSynchronize());

  static constexpr int num_sms = 128;
  static constexpr int kThreads = 512;

  dim3 blocks(num_sms, 1);
  dim3 threads(kThreads, 1, 1);

  static constexpr int kSharedMemSize =
      kN * sizeof(DType) * 3 + 2 * sizeof(std::uint64_t) /*mbarrier*/;

  test_bar_sync<kN><<<blocks, threads, kSharedMemSize>>>(
      thrust::raw_pointer_cast(d_a.data()),
      thrust::raw_pointer_cast(d_b.data()),
      thrust::raw_pointer_cast(d_c.data()));
  CudaCheck(hipDeviceSynchronize());

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    return -1;
  }

  printf("\n\nKernel execution completed successfully\n");
  h_c = d_c;
  for (int i = 0; i < h_c.size(); ++i) {
    printf("%.2f, ", ToFloat(h_c[i]));

    if ((i + 1) % 8 == 0) printf("\n");
  }

  // printf("\n\nh_c_ref: \n");
  // for (int i = 0; i < h_c.size(); ++i) {
  //   printf("%.2f, ", ToFloat(h_c_ref[i]));

  //   if ((i + 1) % 8 == 0) printf("\n");
  // }
  // printf("\n");

  return 0;
}
