#include "hip/hip_runtime.h"
#include "kernel.cuh"

void printResult(const float *mat, size_t height, size_t width) {
  printf("matrix : (%d, %d)\n", height, width);
  for (size_t i = 0; i < height; ++i) {
    for (size_t j = 0; j < width; ++j)
      printf("%f ", mat[i * width + j]);
    printf("\n");
  }
  printf("\n");
}

int main(int argc, char *argv[]) {
  const size_t kMatHeight = 2;
  const size_t kMatWidth = 317;
  const size_t mat_size = kMatHeight * kMatWidth;

  srand(0);

  float *h_a, *h_b;

  hipHostMalloc((void **)&h_a, sizeof(float) * mat_size);
  hipHostMalloc((void **)&h_b, sizeof(float) * mat_size);

  // random initialization of matrix A.
  for (size_t i = 0; i < mat_size; ++i)
    h_a[i] = ((float)rand()) / (float)RAND_MAX;

  // initialize memory that stores computation result to all zeros;
  memset(h_b, 0., sizeof(float) * mat_size);

  // events to count the execution time.
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate memory space on the device.
  float *d_a, *d_b;
  hipMalloc((void **)&d_a, sizeof(float) * mat_size);
  hipMalloc((void **)&d_b, sizeof(float) * mat_size);

  // copy matrix A from host to device memory
  CHECK(hipMemcpy(d_a, h_a, sizeof(float) * mat_size, hipMemcpyHostToDevice));

  // start to count execution time. use the default stream.
  hipEventRecord(start);
  int block_num =
    kMatWidth > 512
    ? 512
    : pow(2, static_cast<int>(log2(static_cast<float>(kMatWidth))));

  dim3 block(block_num, 1);
  dim3 grid(kMatHeight, 1);

  KeMatrixSoftMax<<<grid, block, 0>>>(d_b, d_a, kMatWidth);

  hipEventRecord(stop);
  CHECK(hipEventSynchronize(stop));
  CHECK(hipMemcpy(h_b, d_b, sizeof(float) * mat_size, hipMemcpyDeviceToHost));

  float kernel_elapsed_time;
  hipEventElapsedTime(&kernel_elapsed_time, start, stop);
  printf("kernel execution time elapse : %f\n", kernel_elapsed_time);

  printResult(h_b, kMatHeight, kMatWidth);

  hipFree(d_a);
  hipFree(d_b);
  hipHostFree(h_a);
  hipHostFree(h_b);

  return 0;
}
